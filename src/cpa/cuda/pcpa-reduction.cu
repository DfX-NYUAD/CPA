#include "hip/hip_runtime.h"
/* Copyright (c) 2013 Tescase
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
 * THE SOFTWARE.
 */

#include "pcpa-reduction.hpp"

__global__
void pcpa_reduce(float* data, float* power_pts, unsigned int trace_start, unsigned int num_pts, unsigned int num_traces)
{
	
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	int size = blockDim.x;
	
	extern __shared__ float sdata[];
	
	
	sdata[tid] = -1.0f * data[bid * num_pts + tid + trace_start];
	__syncthreads();


	for (unsigned int s = size / 2; s > 0; s >>=1)
	{
		if (tid < s)
		{
			if (sdata[tid] < sdata[tid + s]) sdata[tid] = sdata[tid + s];
		}
		__syncthreads();
	}


	if(tid == 0) 
	{
		if (!(num_pts%2))
			if (sdata[0] < sdata[num_pts - 1])
				sdata[0] = sdata[num_pts - 1];		
 	
		power_pts[bid] = sdata[0];
	}
}


void pcpa_reduce_wrapper(float *data, float* power_pts, unsigned int trace_start, unsigned int num_pts, unsigned int num_traces)
{
	unsigned int sdata_size = num_pts - trace_start;

	pcpa_reduce<<<num_traces, sdata_size, sdata_size * sizeof(float)>>>(  data,
									power_pts,
									trace_start,
									num_pts,
									num_traces);
}

