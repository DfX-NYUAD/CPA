#include "hip/hip_runtime.h"
/* Copyright (c) 2013 Tescase
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
 * THE SOFTWARE.
 */

#include "pcpa-kernel.hpp"

__global__
void pcpa(unsigned char* ciphertext, float* power_pts, unsigned char* hamming_pts, unsigned int num_pts, unsigned int num_traces, unsigned char* inv_sbox)
{
	
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	
	unsigned char key_byte;
	unsigned char pre_byte;
	unsigned char post_byte;
	unsigned char x;

	int post_row;
	int pre_row;
	int post_col;
	int pre_col;
	int ham_dist;
	int byte_id;

	const int num_bytes = 16;
	const int num_keys = 256;
	

	__shared__ unsigned char cipher[4][4];

	
	cipher[tid%4][tid/4] = ciphertext[(bid * num_bytes) + (tid/4) * 4 + (tid%4)];

	__syncthreads();


	post_row = tid / 4;
	post_col = tid % 4;

	post_byte = cipher[post_row][post_col];

	for (int k = 0; k < num_keys; k++)
	{

		key_byte = (unsigned char)k;

		//aes::shift_rows(post_row, post_col, pre_row, pre_col);
		pre_row = post_row;
		pre_col = post_col - post_row;
		if (pre_col < 0)
			pre_col += 4;

		pre_byte = cipher[pre_row][pre_col];

		//pre_byte = aes::add_round_key(key_byte, pre_byte);
		pre_byte = key_byte ^ pre_byte;

		//pre_byte = aes::inv_sub_bytes(pre_byte);
		pre_byte = inv_sbox[(int)pre_byte];

		byte_id = pre_col * 4 + pre_row;

		//hamming_pts[byte_id][k][i] = pm::hamming_dist(pre_byte, post_byte, 8);
		x = pre_byte ^ post_byte;

		ham_dist = 0;

		while(x)
		{
			ham_dist += x & 1;
			x >>= 1;
		}
		
		hamming_pts[(byte_id * num_keys * num_traces) + (k * num_traces) + bid] = ham_dist;
	}

}
	
void pcpa_wrapper(int gridSize, int blockSize, unsigned char* ciphertext, float* power_pts, unsigned char* hamming_pts, unsigned int num_pts, unsigned int num_traces, unsigned char* inv_sbox)
{
	pcpa<<<gridSize, blockSize>>>(	ciphertext, 
					power_pts, 
					hamming_pts, 
					num_pts, 
					num_traces, 
					inv_sbox);
}	
